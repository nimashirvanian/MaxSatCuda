#include "hip/hip_runtime.h"


#include "hip/hip_runtime.h"
#include ""
#include <hiprand.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <time.h>
#include <random>
#include <iostream>
#include <math.h>
#include <vector>

#include "MaxSatStructures.h"
#include "MaxSatSolvers.h"
#include "MaxSatTests.h"
#include "sortingNetworksNvidia\sortingNetworks_common.h"

using namespace std;

typedef vector< pair<int, bool> > vib;
typedef vector<bool> vb;
typedef vector<int> vi;

#define L(a) (int)((a).size())
#define all(a) (a).begin(), (a).end()
#define mp make_pair

#define Trace(X) cerr << #X << " = " << X << endl
#define _ << " _ " << 


int main()
{
	//sumOrCountFanTest();
	//classInCudaTest();
	//sortTest();
	hipDeviceReset();
	int nbvars, nbclauses;
	cin >> nbvars >> nbclauses;
	nbvars++;
	Cnf* cnf;
	hipMallocManaged(&cnf, sizeof(Cnf));
	new(cnf) Cnf(nbvars);
	while (nbclauses--) {
		int tempvar;
		Clause *tmpcls = new Clause();
		cin >> tempvar;
		while (tempvar) {
			int id = abs(tempvar);
			bool sgn = tempvar > 0;
			tmpcls->addLiteral(id, sgn);
			cin >> tempvar;
		}
		cnf->addClause(*tmpcls);
	}

	//SatSolver *solver = new CudaSingleStepSASatSolver(nbvars, cnf);
	SatSolver *solver = new GreedySatSolver(nbvars, cnf);
	
	solver->solve();
	hipDeviceReset();
    return 0;
}


